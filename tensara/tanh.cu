//Problem Description
//
// Perform the Tanh activation function on an input matrix:
// $$
// C[i][j] = \text{tanh}(A[i][j])
// $$
//
// The Tanh function is defined as:
// $$
// \text{tanh}(x) = \frac{e^x - e^{-x}}{e^x + e^{-x}}
// $$
//
// ## Input:
// - Matrix $A$ of size $M \times N$ containing floating-point values
//
// ## Output:
// - Matrix $C$ of size $M \times N$ containing the Tanh activation values
//
// ## Notes:
// - Both matrices $\text{A}$ and $\text{C}$ are stored in row-major order
// - This problem is adapted from [KernelBench](https://github.com/ScalingIntelligence/KernelBench/blob/main/KernelBench/level1/22_Tanh.py)


#include <hip/hip_runtime.h>

__global__ void kernel(const float* input, float* output, size_t n, size_t m) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if(y < m && x < n) {
        output[y * n + x] = tanhf(input[y * n + x]);
    }
}

__global__ void kernel_faster(const float* input, float* output, size_t n, size_t m) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if(y < m && x < n) {
        output[y * n + x] = tanhf(input[y * n + x]);
    }
}

// Note: input, output are all device pointers to float16 arrays
extern "C" void solution(const float* input, float* output, size_t n, size_t m) {
    dim3 blockDim(16, 16);
    dim3 gridDim((n + blockDim.x - 1) / blockDim.x,
                 (m + blockDim.y - 1) / blockDim.y);
    kernel<<<gridDim, blockDim>>>(input, output, n, m);
    hipDeviceSynchronize();
}
